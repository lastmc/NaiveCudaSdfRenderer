#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <numeric>
#include <algorithm>
#include <chrono>

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"
#include "cuda_device_math.h"

static constexpr auto width = 1280u;
static constexpr auto height = 720u;

static constexpr auto max_ray_depth = 6;
static constexpr auto eps = 1e-4f;
static constexpr auto inf = 1e10f;
static constexpr auto fov = 0.23f;
static constexpr auto dist_limit = 100.0f;
static constexpr auto camera_pos = lc_make_float3(0.0f, 0.32f, 3.7f);
static constexpr auto light_pos = lc_make_float3(-1.5f, 0.6f, 0.3f);
static constexpr auto light_normal = lc_make_float3(1.0f, 0.0f, 0.0f);
static constexpr auto light_radius = 2.0f;

__device__ lc_float intersect_light(lc_float3 pos, lc_float3 d) {
    auto cos_w = lc_dot(-d, light_normal);
    auto dist = lc_dot(d, light_pos - pos);
    auto D = dist / cos_w;
    auto dist_to_center = lc_distance_squared(light_pos, pos + D * d);
    auto valid = cos_w > 0.0f & dist > 0.0f & dist_to_center < light_radius * light_radius;
    return lc_select(inf, D, valid);
}

__device__ lc_uint tea(lc_uint v0, lc_uint v1) {
    lc_uint s0 = 0u;
    for (auto n = 0u; n < 4u; n++) {
        s0 += 0x9e3779b9u;
        v0 += ((v1 << 4) + 0xa341316cu) ^ (v1 + s0) ^ ((v1 >> 5u) + 0xc8013ea4u);
        v1 += ((v0 << 4) + 0xad90777du) ^ (v0 + s0) ^ ((v0 >> 5u) + 0x7e95761eu);
    }
    return v0;
}

__device__ lc_float rand(lc_uint &state) {
    constexpr auto lcg_a = 1664525u;
    constexpr auto lcg_c = 1013904223u;
    state = lcg_a * state + lcg_c;
    // TODO: Probably cast problem
    return static_cast<lc_float>(state & 0x00ffffffu) * (1.0f / static_cast<float>(0x01000000u));
}

__device__ lc_float3 out_dir(lc_float3 n, lc_uint &seed) {
    auto u = lc_select(
        lc_make_float3(1.f, 0.f, 0.f),
        lc_normalize(lc_cross(n, lc_make_float3(0.0f, 1.0f, 0.0f))),
        abs(n.y) < 1.0f - eps);
    auto v = lc_cross(n, u);
    auto phi = 2.0f * 3.1415926f * rand(seed);
    auto ay = sqrt(rand(seed));
    auto ax = sqrt(1.0f - ay * ay);
    return ax * (cos(phi) * u + sin(phi) * v) + ay * n;
};

__device__ lc_float make_nested(lc_float f) {
    static constexpr auto freq = 40.0f;
    f *= freq;
    f = lc_select(f, lc_select(lc_fract(f), 1.f - lc_fract(f), static_cast<int>(f) % 2 == 0), f < 0.f);
    return (f - 0.2f) * (1.0f / freq);
}

__device__ lc_float sdf(lc_float3 o) {
    auto wall = lc_min(o.y + 0.1f, o.z + 0.4f);
    auto sphere = lc_distance(o, lc_make_float3(0.0f, 0.35f, 0.0f)) - 0.36f;
    auto q = lc_abs(o - lc_make_float3(0.8f, 0.3f, 0.0f)) - 0.3f;
    auto box = lc_length(lc_max(q, lc_make_float3(0.0f))) + lc_min(lc_max(lc_max(q.x, q.y), q.z), 0.0f);
    auto O = o - lc_make_float3(-0.8f, 0.3f, 0.0f);
    auto d = lc_make_float2(lc_length(lc_make_float2(O.x, O.z)) - 0.3f, lc_abs(O.y) - 0.3f);
    auto cylinder = lc_min(lc_max(d.x, d.y), 0.0f) + lc_length(lc_max(d, lc_make_float2(0.0f)));
    auto geometry = make_nested(lc_min(lc_min(sphere, box), cylinder));
    auto g = lc_max(geometry, -(0.32f - (o.y * 0.6f + o.z * 0.8f)));
    return lc_min(wall, g);
};

__device__ lc_float ray_march(lc_float3 p, lc_float3 d) {
    auto dist = 0.0f;
    for(auto j = 0; j < 100; j++) {
        auto s = sdf(p + dist * d);
        if(s <= 1e-6f || dist >= inf) { break; };
        dist += s;
    };
    return lc_min(dist, inf);
}

__device__ lc_float3 sdf_normal(lc_float3 p) {
    static constexpr auto d = 1e-3f;
    auto n = lc_make_float3();
    auto sdf_center = sdf(p);
    for (auto i = 0; i < 3; i++) {
        auto inc = p;
        inc[i] += d;
        n[i] = (1.0f / d) * (sdf(inc) - sdf_center);
    }
    return lc_normalize(n);
}

__device__ void next_hit(lc_float &closest, lc_float3 &normal, lc_float3 &c, lc_float3 pos, lc_float3 d) {
    closest = inf;
    normal = lc_make_float3();
    c = lc_make_float3();
    auto ray_march_dist = ray_march(pos, d);
    if(ray_march_dist < lc_min(dist_limit, closest)) {
        closest = ray_march_dist;
        auto hit_pos = pos + d * closest;
        normal = sdf_normal(hit_pos);
        auto t = static_cast<int>((hit_pos.x + 10.0f) * 1.1f + 0.5f) % 3;
        c = lc_make_float3(0.4f) + lc_make_float3(0.3f, 0.2f, 0.3f) * lc_select(lc_make_float3(0.0f), lc_make_float3(1.0f), t == lc_make_int3(0, 1, 2));
    }
}

__global__ void render_kernel(lc_uint* seed_image, lc_float4* accum_image, lc_uint frame_index) {
    // set_block_size(16u, 8u, 1u);

    auto resolution = lc_make_float2(gridDim.x * blockDim.x, gridDim.y * blockDim.y);
    auto coord = lc_make_uint2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
    auto global_id = coord.x + coord.y * gridDim.x * blockDim.x;

    if(frame_index == 0u) {
        seed_image[global_id] = tea(coord.x, coord.y);
        accum_image[global_id] = lc_make_float4(lc_make_float3(0.0f), 1.0f);
    }

    auto aspect_ratio = resolution.x / resolution.y;
    auto pos = camera_pos;
    auto seed = seed_image[global_id];
    auto ux = rand(seed);
    auto uy = rand(seed);
    auto uv = lc_make_float2(coord.x + ux, resolution.y - 1u - coord.y + uy);
    auto d = lc_make_float3(
        2.0f * fov * uv / resolution.y - fov * lc_make_float2(aspect_ratio, 1.0f) - 1e-5f, -1.0f);
    d = lc_normalize(d);
    auto throughput = lc_make_float3(1.0f, 1.0f, 1.0f);
    auto hit_light = 0.0f;
    for(auto depth = 0; depth < max_ray_depth; depth++) {
        auto closest = 0.0f;
        auto normal = lc_make_float3();
        auto c = lc_make_float3();
        next_hit(closest, normal, c, pos, d);
        // accum_image[global_id] = lc_make_float4(uv/500.0, 1.0 , 1.0);
        // return;
        auto dist_to_light = intersect_light(pos, d);
        if(dist_to_light < closest) {
            hit_light = 1.0f;
            break;
        }
        if(lc_length_squared(normal) == 0.0f) { break; };
        auto hit_pos = pos + closest * d;
        d = out_dir(normal, seed);
        pos = hit_pos + 1e-4f * d;
        throughput *= c;
    }
    auto accum = accum_image[global_id];
    auto accum_color = lc_make_float3(accum.x, accum.y, accum.z)
        + lc_make_float3(throughput.x, throughput.y, throughput.z) * hit_light;
    accum_image[global_id] = lc_make_float4(accum_color, 1.0f);
    seed_image[global_id] = seed;
}

int main() {

    std::vector<float> pixels(width * height * 4u);
    std::fill(pixels.begin(), pixels.end(), 1.f);

    lc_uint* seedImage;
    lc_float4* accumImage;
    hipMalloc((void**)&seedImage, sizeof(lc_uint) * width * height);
    hipMalloc((void**)&accumImage, sizeof(lc_float4) * width * height);

    auto tick = std::chrono::high_resolution_clock::now();

    static constexpr auto totalSpp = 2048;
    dim3 block = make_uint3(16, 8, 1);
    dim3 grid = make_uint3(width/16, height/8, 1);
    for(auto spp = 0; spp < totalSpp; spp++){
        render_kernel<<<grid, block>>>(seedImage, accumImage, spp);
    }

    hipMemcpy(pixels.data(), accumImage, sizeof(lc_float4) * width * height, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    auto toc = std::chrono::high_resolution_clock::now();

    using namespace std::chrono_literals;
    printf("Speed = %.2f spp/s\n", totalSpp / ((toc - tick) / 1ns * 1e-6) * 1000);

    for(auto& k: pixels) k /= totalSpp + 1;

    float mean = 0.f;
    for(auto& k: pixels) mean += k;
    mean /= width * height * 4;

    for(auto& k: pixels) k = (k / mean * 0.24);

    stbi_write_hdr("render.hdr", width, height, 4, pixels.data());

}
